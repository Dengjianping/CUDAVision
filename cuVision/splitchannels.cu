#include "hip/hip_runtime.h"
#include "cumath.cuh"

__global__ void split(uchar3 *d_input, int height, int width, size_t pitch, uchar *r_ch, uchar *g_ch, uchar *b_ch) {

}

extern "C"
void cudaSplit(cv::Mat & input, std::vector<cv::Mat> & channels) {
    if (input.channels == 1) {
        channels.push_back(input);
        return;
    }
    channels = std::vector<cv::Mat>(input.channels);

    uchar3 *d_input; uchar *r_ch, *g_ch, *b_ch;
    size_t pitch;
    hipMallocPitch(&d_input, &pitch, sizeof(uchar3)*input.cols, input.rows);
    hipMallocPitch(&r_ch, &pitch, sizeof(uchar)*input.cols, input.rows);
    hipMallocPitch(&g_ch, &pitch, sizeof(uchar)*input.cols, input.rows);
    hipMallocPitch(&b_ch, &pitch, sizeof(uchar)*input.cols, input.rows);

    const int N = 4;
    hipStream_t streams[N];
    for (size_t i = 0; i < N; i++)hipStreamCreate(&streams[i]);
    hipMemcpy2DAsync(d_input, pitch, input.data, sizeof(uchar3)*input.cols, sizeof(uchar3)*input.cols, input.rows, hipMemcpyHostToDevice, streams[0]);
    hipMemcpy2DAsync(r_ch, pitch, input.data, sizeof(uchar)*input.cols, sizeof(uchar)*input.cols, input.rows, hipMemcpyHostToDevice, streams[1]);
    hipMemcpy2DAsync(g_ch, pitch, input.data, sizeof(uchar)*input.cols, sizeof(uchar)*input.cols, input.rows, hipMemcpyHostToDevice, streams[2]);
    hipMemcpy2DAsync(b_ch, pitch, input.data, sizeof(uchar)*input.cols, sizeof(uchar)*input.cols, input.rows, hipMemcpyHostToDevice, streams[3]);

    for (size_t i = 0; i < N; i++)hipStreamSynchronize(streams[i]);

    hipMemset2D(r_ch, pitch, 0, sizeof(uchar)*input.cols, input.rows);
    hipMemset2D(g_ch, pitch, 0, sizeof(uchar)*input.cols, input.rows);
    hipMemset2D(b_ch, pitch, 0, sizeof(uchar)*input.cols, input.rows);
}