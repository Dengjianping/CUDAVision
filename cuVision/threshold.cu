#include "hip/hip_runtime.h"
#include "headers.h"

__global__ void threshold(uchar *input, size_t inputPitch, int imageRows, int imageCols, uchar *output, size_t outputPitch, uchar thresholdValue) {
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    if (row < imageRows&&col < imageCols) {
        uchar *pixelValue = (uchar*)((char*)input + row*inputPitch) + col;
        uchar *outputPixelValue = (uchar*)((char*)output + row*outputPitch) + col;
        if (*pixelValue < thresholdValue) {
            *outputPixelValue = 0;
        }
        else {
            *outputPixelValue = 255;
        }
    }
}

extern "C"
hipError_t cudaThreshold(cv::Mat & input, cv::Mat & output, uchar thresholdValue) {
    output = cv::Mat(input.size(), CV_8U, cv::Scalar(0));
    hipError_t cudaStatus;

    // make sure the image is a gray image
    if (input.type() != CV_8U) {
        cv::cvtColor(input, input, CV_8U);
    }

    uchar *d_input, *d_output;
    size_t inputPitch, outputPitch;

    // get pitch
    cudaStatus = hipMallocPitch(&d_input, &inputPitch, sizeof(uchar)*input.cols, input.rows);
    cudaStatus = hipMallocPitch(&d_output, &outputPitch, sizeof(uchar)*output.cols, output.rows);

    // use stream to accelerate copy operation
    hipStream_t inputCopy, outputCopy;
    cudaStatus = hipStreamCreate(&inputCopy); cudaStatus = hipStreamCreate(&outputCopy);

    // copy data to device
    cudaStatus = hipMemcpy2DAsync(d_input, inputPitch, input.data, sizeof(uchar)*input.cols, sizeof(uchar)*input.cols, input.rows, hipMemcpyHostToDevice, inputCopy);
    cudaStatus = hipMemcpy2DAsync(d_output, outputPitch, output.data, sizeof(uchar)*output.cols, sizeof(uchar)*output.cols, output.rows, hipMemcpyHostToDevice, outputCopy);

    // block until data copy is complete
    cudaStatus = hipStreamSynchronize(inputCopy); cudaStatus = hipStreamSynchronize(outputCopy);

    // define block size and
    dim3 blockSize(input.cols / MAX_THREADS + 1, input.rows / MAX_THREADS + 1);
    dim3 threadSize(MAX_THREADS, MAX_THREADS);

    threshold <<<blockSize, threadSize>>> (d_input, inputPitch, input.rows, input.cols, d_output, outputPitch, thresholdValue);
    cudaStatus = hipDeviceSynchronize();

    cudaStatus = hipMemcpy2D(output.data, sizeof(uchar)*output.cols, d_output, outputPitch, sizeof(uchar)*output.cols, output.rows, hipMemcpyDeviceToHost);

    // resource releasing
    hipStreamDestroy(inputCopy); hipStreamDestroy(outputCopy);
    hipFree(d_input); hipFree(d_output);

    return cudaStatus;
}