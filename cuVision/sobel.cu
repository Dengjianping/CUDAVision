#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include <opencv2\core.hpp>
#include <opencv2\highgui.hpp>
#include <opencv2\imgproc.hpp>

#define CUDA_CALL(x) {const hipError_t a = (x); if (a != hipSuccess) { std::cout << std::endl << "CUDA Error: " << hipGetErrorString(a) << ", error number: " << a << std::endl; hipDeviceReset(); assert(0);}}
#define MAX_THREADS 32
#define K_SIZE 3

__constant__ char sobelKernelXC[K_SIZE][K_SIZE] = { { -1.0,0.0,1.0 },{ -2.0,0.0,2.0 },{ -1.0,0.0,1.0 } };
__constant__ char sobelKernelYC[K_SIZE][K_SIZE] = { { -1.0,-2.0,-1.0 },{ 0.0,0.0,0.0 },{ 1.0,2.0,1.0 } };

__global__ void sobel(uchar *input, int rows, int cols, size_t inputPitch, uchar *gx, size_t gxPitch, uchar *gy, size_t gyPitch, uchar *output, size_t outputPitch) {
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    if (row < rows && col < cols) {
        // convolving
        for (size_t i = 0; i < K_SIZE; i++)
            for (size_t j = 0; j < K_SIZE; j++) {
                uchar *inputValue = (uchar*)((char*)input + row*inputPitch) + col;
                // convolving gx
                uchar *gxValue = (uchar*)((char*)gx + (row + i - K_SIZE / 2)*outputPitch) + (col + j - K_SIZE / 2);
                *gxValue += sobelKernelXC[i][j] * (*inputValue);

                // convolving gy
                uchar *gyValue = (uchar*)((char*)gy + (row + i - K_SIZE / 2)*outputPitch) + (col + j - K_SIZE / 2);
                *gyValue += sobelKernelYC[i][j] * (*inputValue);
        }

        uchar *gxValue = (uchar*)((char*)gx + row*outputPitch) + col;
        uchar *gyValue = (uchar*)((char*)gy + row*outputPitch) + col;
        uchar *outputValue = (uchar*)((char*)output + row*outputPitch) + col;
        *outputValue = *gxValue + *gyValue;
    }
}

extern "C"
void cudaSobel(cv::Mat & input, cv::Mat & output) {
    output = cv::Mat(input.size(), CV_8U, cv::Scalar(0));

    uchar *d_input, *d_output, *gx, *gy;
    size_t inputPitch, outputPitch, gxPitch, gyPitch;

    hipStream_t inputStream, outputStream, gxStream, gyStream;
    CUDA_CALL(hipStreamCreate(&inputStream)); CUDA_CALL(hipStreamCreate(&outputStream)); CUDA_CALL(hipStreamCreate(&gxStream)); CUDA_CALL(hipStreamCreate(&gyStream));

    CUDA_CALL(hipMallocPitch(&d_input, &inputPitch, sizeof(uchar)*input.cols, input.rows));
    CUDA_CALL(hipMallocPitch(&d_output, &outputPitch, sizeof(uchar)*output.cols, output.rows));
    CUDA_CALL(hipMallocPitch(&gx, &gxPitch, sizeof(uchar)*output.cols, output.rows));
    CUDA_CALL(hipMallocPitch(&gy, &gyPitch, sizeof(uchar)*output.cols, output.rows));

    CUDA_CALL(hipMemcpy2DAsync(d_input, inputPitch, input.data, sizeof(uchar)*input.cols, sizeof(uchar)*input.cols, input.rows, hipMemcpyHostToDevice, inputStream));
    CUDA_CALL(hipMemcpy2DAsync(d_output, outputPitch, output.data, sizeof(uchar)*output.cols, sizeof(uchar)*output.cols, output.rows, hipMemcpyHostToDevice, outputStream));
    CUDA_CALL(hipMemcpy2DAsync(gx, gxPitch, output.data, sizeof(uchar)*output.cols, sizeof(uchar)*output.cols, output.rows, hipMemcpyHostToDevice, gxStream));
    CUDA_CALL(hipMemcpy2DAsync(gy, gyPitch, output.data, sizeof(uchar)*output.cols, sizeof(uchar)*output.cols, output.rows, hipMemcpyHostToDevice, gyStream));

    CUDA_CALL(hipStreamSynchronize(inputStream)); CUDA_CALL(hipStreamSynchronize(outputStream)); CUDA_CALL(hipStreamSynchronize(gxStream)); CUDA_CALL(hipStreamSynchronize(gyStream));

    hipMemset(gx, 0, sizeof(uchar)*output.rows*output.cols);
    hipMemset(gy, 0, sizeof(uchar)*output.rows*output.cols);

    /*
    my sample image size is 600 * 450, so we need 600 * 450 threads to process this image on device at least,
    each block can contain 1024 threads at most in my device, so ,I can define block size as 600 * 450 / 1024 = 263 (20 * 15)
    */
    dim3 blockSize(input.cols / MAX_THREADS + 1, input.rows / MAX_THREADS + 1);
    dim3 threadSize(MAX_THREADS, MAX_THREADS);

    sobel<<<blockSize, threadSize>>>(d_input, input.rows, input.cols, inputPitch, gx, gxPitch, gy, gyPitch, d_output, outputPitch);
    CUDA_CALL(hipDeviceSynchronize());

    // get data back
    CUDA_CALL(hipMemcpy2D(output.data, sizeof(uchar)*output.cols, d_output, outputPitch, sizeof(uchar)*output.cols, output.rows, hipMemcpyDeviceToHost));

    // resource releasing
    hipFree(d_input); hipFree(d_output); hipFree(gx); hipFree(gy);
    CUDA_CALL(hipStreamDestroy(inputStream)); CUDA_CALL(hipStreamDestroy(outputStream)); CUDA_CALL(hipStreamDestroy(gxStream)); CUDA_CALL(hipStreamDestroy(gyStream));
}