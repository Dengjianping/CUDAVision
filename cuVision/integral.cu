#include "hip/hip_runtime.h"
#include "cumath.cuh"

__global__ void integral(uchar *d_input, int height, int width, size_t in_pitch, float *d_output, size_t out_pitch, int *max, int *min) {
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    if (row < height  &&  col < width) {
        uchar *in_pixel = (uchar*)((char*)d_input + row*in_pitch) + col;
        float *out_pixel = (float*)((char*)d_output + row*out_pitch) + col;
        float *out_pixel1 = (float*)((char*)d_output + (row-1)*out_pitch) + col;
        float *out_pixel2 = (float*)((char*)d_output + row*out_pitch) + (col-1);
        float *out_pixel3 = (float*)((char*)d_output + (row - 1)*out_pitch) + (col - 1);

        *out_pixel = (float)*in_pixel + *out_pixel1 + *out_pixel2 - (*out_pixel3);
        atomicMin(min, *out_pixel);
        atomicMax(max, *out_pixel);
        //atomicAdd(sum, *out_pixel);
    }
    /*if (row < height  &&  col < width) {
        float *out_pixel = (float*)((char*)d_output + row*out_pitch) + col;
        *out_pixel = (*out_pixel - (float)(*min)) / ((float)(*max) - (float)(*min));
    }*/
}

extern "C"
void cudaIntegral(cv::Mat & input, cv::Mat & output) {
    //input.convertTo(input, CV_32F);
    output = cv::Mat(cv::Size(input.cols + 1, input.rows + 1), CV_32F, cv::Scalar(0));

    uchar *d_input; float *d_output;
    size_t in_pitch, out_pitch;
    hipMallocPitch(&d_input, &in_pitch, sizeof(uchar)*input.cols, input.rows);
    hipMallocPitch(&d_output, &out_pitch, sizeof(float)*output.cols, output.rows);

    int *min, *max;
    hipMalloc(&min, sizeof(int));
    hipMalloc(&max, sizeof(int));
    hipMemset(min, 0, sizeof(int));
    hipMemset(max, 0, sizeof(int));

    hipStream_t in_stream, out_stream;
    hipStreamCreate(&in_stream); hipStreamCreate(&out_stream);

    hipMemcpy2DAsync(d_input, in_pitch, input.data, sizeof(uchar)*input.cols, sizeof(uchar)*input.cols, input.rows, hipMemcpyHostToDevice, in_stream);
    hipMemcpy2DAsync(d_output, in_pitch, output.data, sizeof(float)*output.cols, sizeof(float)*output.cols, output.rows, hipMemcpyHostToDevice, out_stream);

    hipStreamSynchronize(in_stream); hipStreamSynchronize(out_stream);

    dim3 blockSize(input.cols / (MAX_THREADS / 2) + 1, input.rows / MAX_THREADS + 1);
    dim3 threadSize(MAX_THREADS / 2, MAX_THREADS);

    integral <<<blockSize, threadSize>>> (d_input, input.rows, input.cols, in_pitch, d_output,  out_pitch, max, min);
    CUDA_CALL(hipDeviceSynchronize());

    hipMemcpy2D(output.data, sizeof(float)*output.cols, d_output, out_pitch, sizeof(float)*output.cols, output.rows, hipMemcpyDeviceToHost);

    int h_max, h_min;
    hipMemcpy(&h_max, max, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_min, min, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << h_max << std::endl;
    std::cout << h_min << std::endl;

    hipStreamDestroy(in_stream); hipStreamDestroy(out_stream);
    hipFree(d_input); hipFree(d_output); hipFree(max); hipFree(min);

    //input.convertTo(input, CV_8U);
    output.convertTo(output, CV_32S);
}