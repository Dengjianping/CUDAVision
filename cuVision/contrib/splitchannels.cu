#include "hip/hip_runtime.h"
#include "..\cumath\cumath.cuh"

__global__ void split(uchar3 *d_input, int height, int width, size_t in_pitch, uchar *r_ch, uchar *g_ch, uchar *b_ch, size_t out_pitch) {
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    extern __shared__ uchar3 shared[];

    if (row < height && col < width) {
        uchar3 *in_pixel = (uchar3*)((char*)d_input + row*in_pitch) + col;
        shared[threadIdx.y*blockDim.x + threadIdx.x] = *in_pixel;
        __syncthreads();

        uchar *r_pixel = (uchar*)((char*)r_ch + row*out_pitch) + col;
        uchar *g_pixel = (uchar*)((char*)g_ch + row*out_pitch) + col;
        uchar *b_pixel = (uchar*)((char*)b_ch + row*out_pitch) + col;

        *r_pixel = shared[threadIdx.y*blockDim.x + threadIdx.x].x;
        *g_pixel = shared[threadIdx.y*blockDim.x + threadIdx.x].y;
        *b_pixel = shared[threadIdx.y*blockDim.x + threadIdx.x].z;
    }
}

extern "C"
void cudaSplit(cv::Mat & input, std::vector<cv::Mat> & channels) {
    if (input.channels() == 1) {
        channels.push_back(input);
        return;
    }
    //channels = std::vector<cv::Mat>(input.channels);
    for (size_t i = 0; i < input.channels(); i++) {
        cv::Mat ch = cv::Mat(input.size(), CV_8U, cv::Scalar(0));
        channels.push_back(ch);
    }

    uchar3 *d_input; uchar *r_ch, *g_ch, *b_ch;
    size_t in_pitch, out_pitch;
    hipMallocPitch(&d_input, &in_pitch, sizeof(uchar3)*input.cols, input.rows);
    hipMallocPitch(&r_ch, &out_pitch, sizeof(uchar)*input.cols, input.rows);
    hipMallocPitch(&g_ch, &out_pitch, sizeof(uchar)*input.cols, input.rows);
    hipMallocPitch(&b_ch, &out_pitch, sizeof(uchar)*input.cols, input.rows);

    const int N = 4;
    hipStream_t streams[N];
    for (size_t i = 0; i < N; i++)hipStreamCreate(&streams[i]);
    hipMemcpy2DAsync(d_input, in_pitch, input.data, sizeof(uchar3)*input.cols, sizeof(uchar3)*input.cols, input.rows, hipMemcpyHostToDevice, streams[0]);
    hipMemcpy2DAsync(r_ch, out_pitch, channels[0].data, sizeof(uchar)*input.cols, sizeof(uchar)*input.cols, input.rows, hipMemcpyHostToDevice, streams[1]);
    hipMemcpy2DAsync(g_ch, out_pitch, channels[1].data, sizeof(uchar)*input.cols, sizeof(uchar)*input.cols, input.rows, hipMemcpyHostToDevice, streams[2]);
    hipMemcpy2DAsync(b_ch, out_pitch, channels[2].data, sizeof(uchar)*input.cols, sizeof(uchar)*input.cols, input.rows, hipMemcpyHostToDevice, streams[3]);

    for (size_t i = 0; i < N; i++)hipStreamSynchronize(streams[i]);

    //hipMemset2D(r_ch, pitch, 0, sizeof(uchar)*input.cols, input.rows);
    //hipMemset2D(g_ch, pitch, 0, sizeof(uchar)*input.cols, input.rows);
    //hipMemset2D(b_ch, pitch, 0, sizeof(uchar)*input.cols, input.rows);

    dim3 blockSize(input.cols / (MAX_THREADS/2) + 1, input.rows / MAX_THREADS + 1);
    dim3 threadSize(MAX_THREADS/2, MAX_THREADS);
    size_t size = threadSize.x*threadSize.y * sizeof(uchar3);

    split<<<blockSize, threadSize, size>>>(d_input, input.rows, input.cols, in_pitch, r_ch, g_ch, b_ch, out_pitch);
    CUDA_CALL(hipDeviceSynchronize());

    hipMemcpy2DAsync(channels[0].data, sizeof(uchar)*input.cols, r_ch, out_pitch, sizeof(uchar)*input.cols, input.rows, hipMemcpyDeviceToHost, streams[0]);
    hipMemcpy2DAsync(channels[1].data, sizeof(uchar)*input.cols, g_ch, out_pitch, sizeof(uchar)*input.cols, input.rows, hipMemcpyDeviceToHost, streams[1]);
    hipMemcpy2DAsync(channels[2].data, sizeof(uchar)*input.cols, b_ch, out_pitch, sizeof(uchar)*input.cols, input.rows, hipMemcpyDeviceToHost, streams[2]);
    for (size_t i = 0; i < N-1; i++)hipStreamSynchronize(streams[i]);

    for (size_t i = 0; i < N; i++)hipStreamDestroy(streams[i]);
    hipFree(r_ch); hipFree(g_ch); hipFree(b_ch); hipFree(d_input);
}