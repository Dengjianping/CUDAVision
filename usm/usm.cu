#include "hip/hip_runtime.h"
#include "..\cumath\cumath.cuh"

struct __align__(16) float3_aligned { float x, y, z; };

__global__ void usmcolor(float3_aligned *d_input, // target data
                         size_t pitch, // width allocated by hipMallocPitch
                         int height, int width, // image rows and cols
                         float3_aligned *d_output, // output
                         int radius, float theta, float weight) 
{
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    extern __shared__ float gaussianKernel[];
    __shared__ float shared_pixels[6*3][32];
    if (row < height && col < width) 
    {
        float3_aligned inputPixel = *((float3_aligned*)((char*)d_input + row*pitch) + col);
        // use share memory to fix global memory coalescing
        shared_pixels[threadIdx.y][threadIdx.x] = inputPixel.x;
        shared_pixels[threadIdx.y + 6][threadIdx.x] = inputPixel.y;
        shared_pixels[threadIdx.y + 12][threadIdx.x] = inputPixel.z;

        // gaussian kernel
        if (threadIdx.x < 2 * radius + 1 && threadIdx.y < 2 * radius + 1)
            gaussianKernel[threadIdx.y*(2 * radius + 1) + threadIdx.x] = twoDimGaussian(col - radius, radius - row, theta);
        __syncthreads();

        // get gaussian blurring data, convolving
        float3_aligned blurredPixel = { 0,0,0 };
        for (int i = -radius; i <= radius; i++)
            for (int j = -radius; j <= radius; j++) 
            {
                blurredPixel.x += gaussianKernel[(radius + i)*(2 * radius + 1) + (radius + j)] * shared_pixels[threadIdx.y][threadIdx.x]; // r channel
                blurredPixel.y += gaussianKernel[(radius + i)*(2 * radius + 1) + (radius + j)] * shared_pixels[threadIdx.y + 6][threadIdx.x]; // g channel
                blurredPixel.z += gaussianKernel[(radius + i)*(2 * radius + 1) + (radius + j)] * shared_pixels[threadIdx.y + 12][threadIdx.x]; // b channel
            }

        float3_aligned *outputPixel = (float3_aligned*)((char*)d_output + row*pitch) + col;
        outputPixel->x = (shared_pixels[threadIdx.y][threadIdx.x] - weight*blurredPixel.x) / (1.0 - weight); // r channel
        outputPixel->y = (shared_pixels[threadIdx.y + 6][threadIdx.x] - weight*blurredPixel.y) / (1.0 - weight); // g channel
        outputPixel->z = (shared_pixels[threadIdx.y + 12][threadIdx.x] - weight*blurredPixel.z) / (1.0 - weight); // b channel
    }
}

extern "C"
void cudaUSMColor(cv::Mat & input, cv::Mat & output, int radius, float theta = 1.0, float weight = 0.6) 
{
    if (input.channels() != 3)
    {
        std::cout << "this image is not a 3-ch image" << std::endl;
        return;
    }

    input.convertTo(input, CV_32FC3);
    output = cv::Mat(input.size(), input.type(), cv::Scalar(0, 0, 0));

    dim3 threadSize(32, 6);
    dim3 blockSize(input.cols / threadSize.x + 1, input.rows / threadSize.y + 1);

    float3_aligned *d_input, *d_output;
    size_t pitch;
    hipStream_t inputStream, outputStream;

    CUDA_CALL(hipMallocPitch(&d_input, &pitch, sizeof(float3_aligned)*input.cols, input.rows));
    CUDA_CALL(hipMallocPitch(&d_output, &pitch, sizeof(float3_aligned)*output.cols, output.rows));

    CUDA_CALL(hipStreamCreate(&inputStream)); CUDA_CALL(hipStreamCreate(&outputStream));
    CUDA_CALL((hipMemcpy2DAsync(d_input, pitch, input.data, sizeof(float3)*input.cols, sizeof(float3)*input.cols, input.rows, hipMemcpyHostToDevice, inputStream)));
    CUDA_CALL((hipMemcpy2DAsync(d_output, pitch, output.data, sizeof(float3)*output.cols, sizeof(float3)*output.cols, output.rows, hipMemcpyHostToDevice, outputStream)));

    int dynamicSize = (2 * radius + 1)*(2 * radius + 1) * sizeof(float);
    usmcolor<<<blockSize, threadSize, dynamicSize>>> (d_input, pitch, input.rows, input.cols, d_output, radius, theta, weight);
    CUDA_CALL(hipDeviceSynchronize());

    CUDA_CALL(hipMemcpy2D(output.data, sizeof(float3)*output.cols, d_output, pitch, sizeof(float3)*output.cols, output.rows, hipMemcpyDeviceToHost));

    hipStreamDestroy(inputStream); hipStreamDestroy(outputStream);
    hipFree(d_input); hipFree(d_output);

    output.convertTo(output, CV_8UC3);
    input.convertTo(input, CV_8UC3);
}