#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>

#include <iostream>

#define CUDA_CALL(x) {const hipError_t a = (x); if (a != hipSuccess) { std::cout << std::endl << "CUDA Error: " << hipGetErrorString(a) << ", error number: " << a << std::endl; hipDeviceReset(); assert(0);}}
#define K_SIZE 3
#define MAX_THREADS 32

__constant__ uchar erodeKernel[K_SIZE][K_SIZE] = { {0,1,0},{1,1,1},{0,1,0} };

__global__ void erode(uchar *input, size_t inputPitch, int inputRow, int inputCol, uchar *output, size_t outputPitch) {
    extern __shared__ uchar kernel[];
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    if (row < inputRow&&col < inputCol) {
        //convolve2D<uchar>(row, col, input, inputRow, inputCol, inputPitch, K_SIZE, kernel, output, outputPitch);
        for (size_t i = 0; i < K_SIZE; i++) 
            for (size_t j = 0; j < K_SIZE; j++) {
                uchar *inputValue = (uchar*)((char*)input + row*inputPitch) + col;
                uchar *outputValue = (uchar*)((char*)output + (row + i)*outputPitch) + (col + j);
                //*outputValue += erodeKernel[i][j] * (*inputValue);
                *outputValue += erodeKernel[i][j] * (*inputValue);
            }
    }
}

extern "C"
void cudaErode(cv::Mat & input, cv::Mat & output, int iteration = 1) {
    output = cv::Mat(cv::Size(input.cols + K_SIZE - 1, input.rows + K_SIZE - 1), CV_8U, cv::Scalar(0));

    if (input.type() != CV_8U) {
        cv::cvtColor(input, input, CV_8U);
    }

    uchar *d_input, *d_output;
    size_t inputPitch, outputPitch;

    CUDA_CALL(hipMallocPitch(&d_input, &inputPitch, sizeof(uchar)*input.cols, input.rows));
    CUDA_CALL(hipMallocPitch(&d_output, &outputPitch, sizeof(uchar)*output.cols, output.rows));

    hipStream_t inputCopy, outputCopy;
    CUDA_CALL(hipStreamCreate(&inputCopy)); CUDA_CALL(hipStreamCreate(&outputCopy));

    CUDA_CALL(hipMemcpy2DAsync(d_input, inputPitch, input.data, sizeof(uchar)*input.cols, sizeof(uchar)*input.cols, input.rows, hipMemcpyHostToDevice, inputCopy));
    CUDA_CALL(hipMemcpy2DAsync(d_output, outputPitch, output.data, sizeof(uchar)*output.cols, sizeof(uchar)*output.cols, output.rows, hipMemcpyHostToDevice, outputCopy));

    CUDA_CALL(hipStreamSynchronize(inputCopy)); CUDA_CALL(hipStreamSynchronize(outputCopy));

    dim3 blockSize(input.cols / MAX_THREADS + 1, input.rows / MAX_THREADS + 1);
    dim3 threadSize(MAX_THREADS, MAX_THREADS);

    int sharedSize = sizeof(uchar)*K_SIZE*K_SIZE;
    erode <<<blockSize, threadSize, sharedSize>>> (d_input, inputPitch, input.rows, input.cols, d_output, outputPitch);
    CUDA_CALL(hipDeviceSynchronize());

    // copy data back to host
    CUDA_CALL(hipMemcpy2D(output.data, sizeof(uchar)*output.cols, d_output, outputPitch, sizeof(uchar)*output.cols, output.rows, hipMemcpyDeviceToHost));

    // resource releasing
    hipStreamDestroy(inputCopy); hipStreamDestroy(outputCopy);
    hipFree(d_input); hipFree(d_output);
}